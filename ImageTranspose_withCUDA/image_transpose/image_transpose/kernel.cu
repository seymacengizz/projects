#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <opencv2/opencv.hpp>
#include <assert.h>
#pragma once
#ifdef __INTELLISENSE__
void __syncthreads();
#endif
using namespace cv;
using namespace std;
using namespace gpu;
// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
	if (result != hipSuccess) {
		fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
		assert(result == hipSuccess);
	}
#endif
	return result;
}

const int TILE_DIM = 32;
const int BLOCK_ROWS = 8;
const int NUM_REPS = 100;

// Check errors and print GB/s
void postprocess(const uchar *ref, const uchar *res, int n, uchar ms)
{
	bool passed = true;
	for (int i = 0; i < n; i++)
		if (res[i] != ref[i]) {
			printf("%d %f %f\n", i, res[i], ref[i]);
			printf("%25s\n", "*** FAILED ***");
			passed = false;
			break;
		}
	if (passed)
		printf("%20.2f\n", 2 * n * sizeof(uchar) * 1e-6 * NUM_REPS / ms);
}

// simple copy kernel
// Used as reference case representing best effective bandwidth.
__global__ void copy(uchar *odata, const uchar *idata)
{
	int x = blockIdx.x * TILE_DIM + threadIdx.x;
	int y = blockIdx.y * TILE_DIM + threadIdx.y;
	int width = gridDim.x * TILE_DIM;

	for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
		odata[(y + j)*width + x] = idata[(y + j)*width + x];
}

// copy kernel using shared memory
// Also used as reference case, demonstrating effect of using shared memory.
__global__ void copySharedMem(uchar *odata, const uchar *idata)
{
	__shared__ uchar tile[TILE_DIM * TILE_DIM];

	int x = blockIdx.x * TILE_DIM + threadIdx.x;
	int y = blockIdx.y * TILE_DIM + threadIdx.y;
	int width = gridDim.x * TILE_DIM;

	for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
		tile[(threadIdx.y + j)*TILE_DIM + threadIdx.x] = idata[(y + j)*width + x];

	__syncthreads();

	for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
		odata[(y + j)*width + x] = tile[(threadIdx.y + j)*TILE_DIM + threadIdx.x];
}

// naive transpose
// Simplest transpose; doesn't use shared memory.
// Global memory reads are coalesced but writes are not.
__global__ void transposeNaive(uchar *odata, const uchar *idata)
{
	int x = blockIdx.x * TILE_DIM + threadIdx.x;
	int y = blockIdx.y * TILE_DIM + threadIdx.y;
	int width = gridDim.x * TILE_DIM;

	for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
		odata[x*width + (y + j)] = idata[(y + j)*width + x];

	//printf("\ndone...\n");
}

// coalesced transpose
// Uses shared memory to achieve coalesing in both reads and writes
// Tile width == #banks causes shared memory bank conflicts.
__global__ void transposeCoalesced(uchar *odata, const uchar *idata)
{
	__shared__ uchar tile[TILE_DIM][TILE_DIM];

	int x = blockIdx.x * TILE_DIM + threadIdx.x;
	int y = blockIdx.y * TILE_DIM + threadIdx.y;
	int width = gridDim.x * TILE_DIM;

	for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
		tile[threadIdx.y + j][threadIdx.x] = idata[(y + j)*width + x];

	__syncthreads();

	x = blockIdx.y * TILE_DIM + threadIdx.x;  // transpose block offset
	y = blockIdx.x * TILE_DIM + threadIdx.y;

	for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
		odata[(y + j)*width + x] = tile[threadIdx.x][threadIdx.y + j];
}


// No bank-conflict transpose
// Same as transposeCoalesced except the first tile dimension is padded 
// to avoid shared memory bank conflicts.
__global__ void transposeNoBankConflicts(uchar *odata, const uchar *idata)
{
	__shared__ uchar tile[TILE_DIM][TILE_DIM + 1];

	int x = blockIdx.x * TILE_DIM + threadIdx.x;
	int y = blockIdx.y * TILE_DIM + threadIdx.y;
	int width = gridDim.x * TILE_DIM;

	for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
		tile[threadIdx.y + j][threadIdx.x] = idata[(y + j)*width + x];

	__syncthreads();

	x = blockIdx.y * TILE_DIM + threadIdx.x;  // transpose block offset
	y = blockIdx.x * TILE_DIM + threadIdx.y;

	for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
		odata[(y + j)*width + x] = tile[threadIdx.x][threadIdx.y + j];

}

int main(int argc, char **argv)
{
	char FullFileName[100];
	char FirstFileName[100] = "C:\\Users\\user\\Desktop\\GPU Computing\\Lena.png";

	int FileNum = 96; //262; 
	printf("%s\n", FirstFileName);
	//read image file  
	Mat img, img_gray, img_gray1;
	Mat newImage;
	img = imread(FirstFileName);
	cvtColor(img, img_gray, CV_RGB2GRAY);
	uchar *data = img_gray.data;


	//GpuMat d_img(img_gray);
	//GpuMat d_outimg;

	const int nx = img_gray.rows;//1024
	const int ny = img_gray.cols;//1024
	const int mem_size = nx*ny*sizeof(uchar);


	dim3 dimGrid(nx / TILE_DIM, ny / TILE_DIM, 1);
	dim3 dimBlock(TILE_DIM, BLOCK_ROWS, 1);

	int devId = 0;
	if (argc > 1) devId = atoi(argv[1]);

	hipDeviceProp_t prop;
	checkCuda(hipGetDeviceProperties(&prop, devId));
	printf("\nDevice : %s\n", prop.name);
	printf("Matrix size: %d %d, Block size: %d %d, Tile size: %d %d\n",
		nx, ny, TILE_DIM, BLOCK_ROWS, TILE_DIM, TILE_DIM);
	printf("dimGrid: %d %d %d. dimBlock: %d %d %d\n",
		dimGrid.x, dimGrid.y, dimGrid.z, dimBlock.x, dimBlock.y, dimBlock.z);

	checkCuda(hipSetDevice(devId));

	//uchar *h_idata = (uchar*)malloc(mem_size);
	uchar *h_cdata = (uchar*)malloc(mem_size);
	uchar *h_tdata = (uchar*)malloc(mem_size);
	uchar *gold = (uchar*)malloc(mem_size);
	Mat outImg, rgb_outImg;
	uchar *d_idata, *d_cdata, *d_tdata;

	checkCuda(hipMalloc(&d_idata, mem_size));
	checkCuda(hipMalloc(&d_cdata, mem_size));
	checkCuda(hipMalloc(&d_tdata, mem_size));

	// check parameters and calculate execution configuration
	if (nx % TILE_DIM || ny % TILE_DIM) {
		printf("nx and ny must be a multiple of TILE_DIM\n");
		goto error_exit;
	}

	if (TILE_DIM % BLOCK_ROWS) {
		printf("TILE_DIM must be a multiple of BLOCK_ROWS\n");
		goto error_exit;
	}

	// host
	/*for (int j = 0; j < ny; j++)
	for (int i = 0; i < nx; i++)
	h_idata[j*nx + i] = j*nx + i;*/

	// correct result for error checking
	for (int j = 0; j < ny; j++)
		for (int i = 0; i < nx; i++)
			gold[j*nx + i] = img_gray.data[i*nx + j];

	//// device
	checkCuda(hipMemcpy(d_idata, img_gray.data, mem_size, hipMemcpyHostToDevice));

	// events for timing
	hipEvent_t startEvent, stopEvent;
	checkCuda(hipEventCreate(&startEvent));
	checkCuda(hipEventCreate(&stopEvent));
	float ms;

	// ------------
	// time kernels
	// ------------
	printf("\n%25s%25s\n", "Routine", "Bandwidth (GB/s)");

	// ----
	// copy 
	// ----
	printf("%25s", "copy");
	checkCuda(hipMemset(d_cdata, 0, mem_size));
	// warm up
	//copy << <dimGrid, dimBlock >> >(d_cdata, d_idata);
	checkCuda(hipEventRecord(startEvent, 0));
	for (int i = 0; i < NUM_REPS; i++)
		copy << <dimGrid, dimBlock >> >(d_cdata, d_idata);
	checkCuda(hipEventRecord(stopEvent, 0));
	checkCuda(hipEventSynchronize(stopEvent));
	checkCuda(hipEventElapsedTime(&ms, startEvent, stopEvent));
	checkCuda(hipMemcpy(h_cdata, d_cdata, mem_size, hipMemcpyDeviceToHost));
	postprocess(img_gray.data, h_cdata, nx*ny, ms);

	/* -------------
	copySharedMem
	-------------*/
	printf("%25s", "shared memory copy");
	checkCuda(hipMemset(d_cdata, 0, mem_size));
	// warm up
	copySharedMem << <dimGrid, dimBlock >> >(d_cdata, d_idata);
	checkCuda(hipEventRecord(startEvent, 0));
	for (int i = 0; i < NUM_REPS; i++)
		copySharedMem << <dimGrid, dimBlock >> >(d_cdata, d_idata);
	checkCuda(hipEventRecord(stopEvent, 0));
	checkCuda(hipEventSynchronize(stopEvent));
	checkCuda(hipEventElapsedTime(&ms, startEvent, stopEvent));
	checkCuda(hipMemcpy(h_cdata, d_cdata, mem_size, hipMemcpyDeviceToHost));
	postprocess(img_gray.data, h_cdata, nx * ny, ms);

	// --------------
	// transposeNaive 
	// --------------
	printf("%25s", "naive transpose");
	checkCuda(hipMemset(d_tdata, 0, mem_size));
	////warmup
	transposeNaive << <dimGrid, dimBlock >> >(d_tdata, d_idata);
	checkCuda(hipEventRecord(startEvent, 0));
	for (int i = 0; i < NUM_REPS; i++)
		transposeNaive << <dimGrid, dimBlock >> >(d_tdata, d_idata);
	checkCuda(hipEventRecord(stopEvent, 0));
	checkCuda(hipEventSynchronize(stopEvent));
	checkCuda(hipEventElapsedTime(&ms, startEvent, stopEvent));
	checkCuda(hipMemcpy(h_tdata, d_tdata, mem_size, hipMemcpyDeviceToHost));
	//outImg = Mat(nx, ny, CV_8UC1, h_tdata);
	////outImg.data = h_tdata;
	////show image  
	//imshow("origin", outImg);
	//waitKey(0);
	//imwrite("C:\\Users\\user\\Desktop\\tImg1.png", outImg);
	postprocess(gold, h_tdata, nx * ny, ms);

	// ------------------
	// transposeCoalesced 
	// ------------------
	printf("%25s", "coalesced transpose");
	checkCuda(hipMemset(d_tdata, 0, mem_size));
	// warmup
	transposeCoalesced << <dimGrid, dimBlock >> >(d_tdata, d_idata);
	checkCuda(hipEventRecord(startEvent, 0));
	for (int i = 0; i < NUM_REPS; i++)
		transposeCoalesced << <dimGrid, dimBlock >> >(d_tdata, d_idata);
	checkCuda(hipEventRecord(stopEvent, 0));
	checkCuda(hipEventSynchronize(stopEvent));
	checkCuda(hipEventElapsedTime(&ms, startEvent, stopEvent));
	checkCuda(hipMemcpy(h_tdata, d_tdata, mem_size, hipMemcpyDeviceToHost));
	outImg = Mat(nx, ny, CV_8UC1, h_tdata);


	postprocess(gold, h_tdata, nx * ny, ms);

	// ------------------------
	// transposeNoBankConflicts
	// ------------------------
	printf("%25s", "conflict-free transpose");
	checkCuda(hipMemset(d_tdata, 0, mem_size));
	// warmup
	transposeNoBankConflicts << <dimGrid, dimBlock >> >(d_tdata, d_idata);
	checkCuda(hipEventRecord(startEvent, 0));
	for (int i = 0; i < NUM_REPS; i++)
		transposeNoBankConflicts << <dimGrid, dimBlock >> >(d_tdata, d_idata);
	checkCuda(hipEventRecord(stopEvent, 0));
	checkCuda(hipEventSynchronize(stopEvent));
	checkCuda(hipEventElapsedTime(&ms, startEvent, stopEvent));
	checkCuda(hipMemcpy(h_tdata, d_tdata, mem_size, hipMemcpyDeviceToHost));
	postprocess(gold, h_tdata, nx * ny, ms);

	//resizing 
	resize(img_gray, img_gray, Size(512, 512));
	resize(outImg, outImg, Size(512, 512));
	hconcat(img_gray, outImg, newImage);
	imshow("Display side by side", newImage);
	waitKey(0);
	//imwrite("C:\\Users\\user\\Desktop\\newImage.png", newImage);
error_exit:
	// cleanup
	checkCuda(hipEventDestroy(startEvent));
	checkCuda(hipEventDestroy(stopEvent));
	checkCuda(hipFree(d_tdata));
	checkCuda(hipFree(d_cdata));
	checkCuda(hipFree(d_idata));
	//free(h_idata);
	free(h_tdata);
	free(h_cdata);
	free(gold);

}